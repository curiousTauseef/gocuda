/* Simple CUDA Example -- Williams */

#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <stdio.h>

// __global__ means this function is available on CPU and GPU


// This version does NOT print any data out for debugging
__global__
void scale(unsigned int n, float *x, float *y)
{
  unsigned int i, base=blockIdx.x*blockDim.x+threadIdx.x, incr=blockDim.x*gridDim.x;
  for (i=base;i<n;i+=incr)  // note that i>=n is discarded
    x[i]=x[i]*y[i];

}


// This version DOES print data out for debugging
__global__
void scaleprint(unsigned int n, float *x, float *y)
{
  unsigned int i, base=blockIdx.x*blockDim.x+threadIdx.x, incr=blockDim.x*gridDim.x;
  printf("t=%x: base=%d incr=%d n=%d block=%d\n",threadIdx.x, base, incr, n,blockIdx.x);	
  for (i=base;i<n;i+=incr)  // note that i>=n is discarded
    x[i]=x[i]*y[i];

}




// Configurations
// N is the data set size
int N=16;  // do not make this unsigned because we use -N => 1<<|N|
// Default block size
unsigned int blksize=512;
// Number of blocks
unsigned int nblock=0;

// Note, if N=blksize*nblock then you will get a grid of 1 and the kernels will each process one data item
// However, if N>blksize*nblock then each kernel will run its loop and process more than one element


// options
int nonorm=0;  // option - 1 means no CPU run (implies notest)
int nocuda=0;  // option - 1 means no GPU run (implies notest)
int notest=0;  // option - 1 means don't compare results
int print=0;   // option - 1 means use printing kernel
int enddebug=0;  // option - 1 means print end debug

int process_options(int,char *[]);


int main(int argc, char *argv[])
{
  float *x;  // input vector
  float *y;  // output vector
  float *testvec;  // vector for testing (CPU computed)

  // read command line
  int rv=process_options(argc, argv);
  if (rv) return rv;
  

// We need shared memory that the CPU and GPU can both access
  std::cout<<"N="<<N<<std::endl;  // allocate shared memory
  hipMallocManaged(&x,N*sizeof(float));
  hipMallocManaged(&y,N*sizeof(float));
  if (nonorm==0) testvec=(float *)malloc(N*sizeof(float));
  
// Generate input vectors
  for (unsigned int i=0;i<N;i++)
    {
      x[i]=1.0f;
      y[i]=(i%10)/10.0f;
// If we are supposed to do a normal run, do it now
      if (nonorm==0)
         testvec[i]=x[i]*y[i];  // compute right answer old fashioned way
    }
  if (nonorm==1) std::cout<<"Normal skipped"<<std::endl;
// Do cuda run unless disabled
  if (nocuda==0)
    {
    int numblk=(N+blksize-1)/blksize; // Number of whole blocks required to contain N
    if (nblock) numblk=nblock;  // override if provided
    std::cout<<"Start kernel "<<blksize<<" "<<numblk<<std::endl;
// This is the line that kicks off the kernel
     if (print)
          scaleprint<<<numblk,blksize>>>((unsigned int)N,x,y);
     else
          scale<<<numblk,blksize>>>((unsigned int)N,x,y);
// Wait for processing to complete
     hipDeviceSynchronize();
    } else std::cout<<"Cuda skipped"<<std::endl;
// Compare results if asked to do so
  if (notest==0)
    {
      for (unsigned int i=0;i<N;i++)
       {
         if (x[i]!=testvec[i])
	  {
	    std::cout<<"Error at "<<i<<std::endl;
	  }
       }
     if (testvec) free(testvec);
     std::cout<<"Test Complete"<<std::endl;     
  }
// Dump some results
  if (enddebug==1)
  {
    std::cout<<"First 16 elements:"<<std::endl;
    for (unsigned int i=0;i<16;i++) std::cout<<(nocuda==0?x[i]:testvec[i])<<" ";
    std::cout<<std::endl;
  }
// release memory
  hipFree(x);
  hipFree(y);
  return 0;
}

      
      
// Process options
int process_options(int argc, char *argv[])
{
// Process options
  while (argc-->1)
    {
    argv++;
      if (**argv=='?')
	{
	  std::cout<<"Usage: gocuda [g|c] [p] [d] [bs=block_size] [nb=num_blocks] [number_of_samples]"<<std::endl;
	  std::cout<<"   g=GPU only; c=CPU only; d=end debugging dump; p=print inside kernel (for gpu mode)"<<std::endl;
	  std::cout<<"   if number of samples is negative, use 2**|number_of_samples|"<<std::endl;
	  std::cout<<"For example, gocuda -4 produces 16 samples (2**4)"<<std::endl;
	  return 1;
	}
      if (**argv=='d')  // print debug output at end
      {
        enddebug=1;
	continue;
      }
      if (**argv=='p')  // using printing kernel
      {
        print=1;
	continue;
      }
      if (**argv=='c') // no CUDA
        {
	  notest=nocuda=1;
	  continue;
        }
      if (**argv=='g')  // no CPU
	{
	  notest=nonorm=1;
	  continue;
	}
      if (argv[0][0]=='b' && argv[0][1]=='s' && argv[0][2]=='=' )  // block size
	{
	  blksize=atoi(*argv+3);
	  if (blksize<=0)
	    {
	      std::cout<<"Error: Blocksize must be positive!"<<std::endl;
	      return 3;
	    }
	  continue;
	}
      if (argv[0][0]=='n' && argv[0][1]=='b' && argv[0][2]=='=' )  // # of blocks
      {
         nblock=atoi(*argv+3);
	 if (nblock==0) std::cout<<"Warning: Number of blocks zero; will be auto calculated"<<std::endl;
	 continue;
      }
      N=atoi(*argv);
      if (N<0&&N>-31) N=1<<-N;   // if N is negative, take N as 1<<|N|
      if (N<=0)
      {
	std::cout<<"Error: N must be non-zero!"<<std::endl;
	return 2;
      }
    }
  return 0;
}


